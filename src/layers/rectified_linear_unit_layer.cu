#include "hip/hip_runtime.h"
/* Copyright (c) 2018 Bartosz Białoskórski

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
==============================================================================*/

#include "layers/rectified_linear_unit_layer.hpp"

#include <iostream>

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"
#include ""
#include "gpu_allocation_manager.hpp"
#include "gpu_utilities.cuh"

namespace neuralnet {

__global__ void rectifier_kernel(double* d_output, double* d_activation,
                                 int num_inputs) {
  // R(x) = max(0, x)
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < num_inputs) {
    double activation = d_activation[tid];
    d_output[tid] = activation > 0.0 ? activation : 0.0;
  }
}

__global__ void compute_relu_error_kernel(double* __restrict__ d_error,
    const double* __restrict__ d_activation,
    const double* __restrict__ d_weighted_error, int size) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < size)
    d_error[tid] = d_activation[tid] > 0.0 ? d_weighted_error[tid] : 0.0;
}

void ReLuLayer::ForwardPropGpu(const std::vector<double>& input) {
  GpuAllocationManager manager;
  int mini_batch_size = input.size() / (num_inputs_ - 1);
  double* d_activation = (double*)manager.AllocateDevice(
      num_neurons_ * mini_batch_size * sizeof(double));
  double* d_output = (double*)manager.AllocateDevice(
      num_neurons_ * mini_batch_size * sizeof(double));
  // Resizing output and activation to fit size of current mini-batch.
  output_.resize(num_neurons_ * mini_batch_size);
  activation_.resize(num_neurons_ * mini_batch_size);

  ComputeActivationGpu(d_activation, input);

  dim3 grid(std::ceil((float)output_.size() / (float)kBlockSize));
  dim3 block(kBlockSize);
  // Applying rectifier to computed activation.
  rectifier_kernel<<<grid, block>>>(d_output, d_activation, input.size());

  // Copying calculated activation and output to host.
  hipError_t cuda_status;
  cuda_status = hipMemcpy(activation_.data(), d_activation,
                           activation_.size() * sizeof(double),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying activation data to host.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(output_.data(), d_output,
                           output_.size() * sizeof(double),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying output data to host.";
    throw std::runtime_error(err_msg);
  }

  // Freeing memory allocated on device.
  manager.FreeDevice(d_activation);
  manager.FreeDevice(d_output);
}

void ReLuLayer::BackPropGpu(const std::vector<double>& weighted_error_top,
                            const std::vector<double>& prev_layer_output,
                            double momentum) {
  if (momentum <= 0 || momentum >= 1)
    throw std::invalid_argument("momentum coefficient should have a value\
 between 0 and 1.");

  GpuAllocationManager manager;
  int mini_batch_size = weighted_error_top.size() / num_neurons_;
  double* d_error = (double*)manager.AllocateDevice(
      num_neurons_ * mini_batch_size * sizeof(double));
  double* d_activation = (double*)manager.AllocateDevice(
      activation_.size() * sizeof(double));
  double* d_weights = (double*)manager.AllocateDevice(
      weights_.size() * sizeof(double));
  double* d_velocity = (double*)manager.AllocateDevice(
      velocity_.size() * sizeof(double));
  double* d_weighted_error_top = (double*)manager.AllocateDevice(
      weighted_error_top.size() * sizeof(double));
  double* d_prev_layer_output = (double*)manager.AllocateDevice(
      prev_layer_output.size() * sizeof(double));
  double* d_weighted_error = (double*)manager.AllocateDevice(
      (num_inputs_ - 1) * mini_batch_size * sizeof(double));

  // Resizing error and weigted_error to fit size of current mini-batch.
  error_.resize(num_neurons_ * mini_batch_size);
  weighted_error_.resize((num_inputs_ - 1) * mini_batch_size);

  hipError_t cuda_status;
  // Moving data to device.
  cuda_status = hipMemcpy(d_activation, activation_.data(),
                           activation_.size() * sizeof(double),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying activation data to device.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(d_weights, weights_.data(),
                           weights_.size() * sizeof(double),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying weights data to device.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(d_velocity, velocity_.data(),
                           velocity_.size() * sizeof(double),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying velocity data to device.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(d_weighted_error_top, weighted_error_top.data(),
                           weighted_error_top.size() * sizeof(double),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying weighted error data to\
 device.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(d_prev_layer_output, prev_layer_output.data(),
                           prev_layer_output.size() * sizeof(double),
                           hipMemcpyHostToDevice);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying previous layer output\
 data to device.";
    throw std::runtime_error(err_msg);
  }

  dim3 grid(std::ceil((float)error_.size() / (float)kBlockSize));
  dim3 block(kBlockSize);

  // Computing layer's error term.
  compute_relu_error_kernel<<<grid, block>>>(d_error, d_activation,
                                             d_weighted_error_top,
                                             error_.size());

  ComputeVelocityGpu(d_velocity, d_error, d_prev_layer_output, momentum);

  ComputeWeightedErrorGpu(d_weighted_error, d_weights, d_error);

  // Moving results to host.
  cuda_status = hipMemcpy(velocity_.data(), d_velocity,
                           velocity_.size() * sizeof(double),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying velocity data to host.";
    throw std::runtime_error(err_msg);
  }

  cuda_status = hipMemcpy(weighted_error_.data(), d_weighted_error,
                           weighted_error_.size() * sizeof(double),
                           hipMemcpyDeviceToHost);
  if (cuda_status != hipSuccess) {
    std::string err_msg = "Error ("
                          + std::string(hipGetErrorString(cuda_status))
                          + ") occured while copying weighted error data to\
 host.";
    throw std::runtime_error(err_msg);
  }

  // Freeing memory allocated on device.
  manager.FreeDevice(d_error);
  manager.FreeDevice(d_activation);
  manager.FreeDevice(d_weights);
  manager.FreeDevice(d_velocity);
  manager.FreeDevice(d_weighted_error_top);
  manager.FreeDevice(d_prev_layer_output);
  manager.FreeDevice(d_weighted_error);
}

}
